#include "hip/hip_runtime.h"
//===================================================================================================================
// File:        main.cpp
// Created on:  27-10-11
// Authors:     Dirk Vos, Mark Schrauwen, Michiel van der Vlag
//
// Description: The filter file called from main.cpp. This file contains the actual CUDA code. Different functions
//				of the code can be enabled\disabled by (un)commenting the macros as defined below.
//===================================================================================================================

#include <Timer.hpp>
#include <iostream>
#include <iomanip>

//-------------------------------------------------------------------------------------------------------------------
// Global definitions, macros
//-------------------------------------------------------------------------------------------------------------------
// Use commenting to disable/enable functions
#define MAX_BLOCKSIZE	32			// The size of one square block. A block will have size MAX_BLOCKSIZE * MAX_BLOCKSIZE
#define DATA_SIZE		7			// The number of variables of Memory Mapping part.
#define SHARED_MEM					// If uncommented, shared memory will be used.
#define TEXTURE_MEM					// If uncommented, Texture memory will be used.

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

// global information
static unsigned int numMultiProcessors;
static unsigned int numThreadsPerBlock[2];

// define 2D textures
texture<unsigned char, hipTextureType2D, hipReadModeElementType> texture2DRed;
texture<unsigned char, hipTextureType2D, hipReadModeElementType> texture2DGreen;
texture<unsigned char, hipTextureType2D, hipReadModeElementType> texture2DBlue;


//-------------------------------------------------------------------------------------------------------------------
// Memory Mapping
//-------------------------------------------------------------------------------------------------------------------
/**
 *   unsigned int *dev_data is filled in the following way:
 *	  index		content
 *		0      	width
 *		1		height
 *		2		HISTOGRAM_SIZE
 *		3		CONTRAST_THRESHOLD
 *		4		pitch red image
 *		5		pitch green image
 *		6		pitch blue image
 *		7		min			(filled on device)
 *		8		max			(filled on device)
 *		9		histogram	(filled on device)
 */


//-------------------------------------------------------------------------------------------------------------------
// Device discovery
//-------------------------------------------------------------------------------------------------------------------
__host__ void getCudaDeviceInformation(void) {
	// get GPU device
	int device;
	if (hipGetDevice(&device) != hipSuccess) {
		cout << "main - cuda get device failed" << endl;
		exit(1);
	}

	// get GPU properties
	hipDeviceProp_t prop;
	if (hipGetDeviceProperties (&prop, device) != hipSuccess) {
		cout << "main - cuda get device properties failed" << endl;
		exit(1);
	}

	// save to global information
	numMultiProcessors    = prop.multiProcessorCount;
	numThreadsPerBlock[0] = MAX_BLOCKSIZE;
	numThreadsPerBlock[1] = MAX_BLOCKSIZE;
}


//-------------------------------------------------------------------------------------------------------------------
// helper functions
//-------------------------------------------------------------------------------------------------------------------
__device__ unsigned char getElementRed(unsigned int x, unsigned int y) {
	return tex2D(texture2DRed, x, y);
}

__device__ unsigned char getElementGreen(unsigned int x, unsigned int y) {
	return tex2D(texture2DGreen, x, y);
}

__device__ unsigned char getElementBlue(unsigned int x, unsigned int y) {
	return tex2D(texture2DBlue, x, y);
}

__device__ unsigned int getWidth(unsigned int *data) {
	return data[0];
}

__device__ unsigned int getHeight(unsigned int *data) {
	return data[1];
}

__device__ unsigned int getHistogramSize(unsigned int *data) {
	return data[2];
}

__device__ unsigned int getContrastThreshold(unsigned int *data) {
	return data[3];
}

__device__ unsigned int getPitchRedImage(unsigned int *data) {
	return data[4];
}

__device__ unsigned int getPitchGreenImage(unsigned int *data) {
	return data[5];
}

__device__ unsigned int getPitchBlueImage(unsigned int *data) {
	return data[6];
}

__device__ unsigned int getMin(unsigned int *data) {
	return data[7];
}

__device__ unsigned int getMax(unsigned int *data) {
	return data[8];
}

__device__ void setMin(unsigned int *data, unsigned int value) {
	data[7] = value;
}

__device__ void setMax(unsigned int *data, unsigned int value) {
	data[8] = value;
}

__device__ unsigned int * getHistogram(unsigned int *data) {
	return &data[9];
}

//-------------------------------------------------------------------------------------------------------------------
// rgb2gray
//-------------------------------------------------------------------------------------------------------------------
#ifdef TEXTURE_MEM
__global__ void rgb2grayCudaKernel(unsigned char *image, unsigned int *data) {
	// load width and height
	unsigned int width  = getWidth(data);
	unsigned int height = getHeight(data);

	// load block ID's. Thread ids
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pitch = getPitchRedImage(data);

#ifdef SHARED_MEM
	// only run threads that are in the image
	if (x < width && y < height) {
		// allocate shared mem for this block
		__shared__ unsigned char redImage[MAX_BLOCKSIZE][MAX_BLOCKSIZE];
		__shared__ unsigned char greenImage[MAX_BLOCKSIZE][MAX_BLOCKSIZE];
		__shared__ unsigned char blueImage[MAX_BLOCKSIZE][MAX_BLOCKSIZE];

		// copy from global mem to shared mem
		redImage[threadIdx.x][threadIdx.y]   = getElementRed(x, y);
		greenImage[threadIdx.x][threadIdx.y] = getElementGreen(x, y);
		blueImage[threadIdx.x][threadIdx.y]  = getElementBlue(x, y);

		__syncthreads();

		// execute grey scaling code
		float grayPix = 0.0f;
		float r       = static_cast< float >(redImage[threadIdx.x][threadIdx.y]);
		float g       = static_cast< float >(greenImage[threadIdx.x][threadIdx.y]);
		float b       = static_cast< float >(blueImage[threadIdx.x][threadIdx.y]);
		grayPix       = (0.3f * r) + (0.59f * g) + (0.11f * b);

		__syncthreads();

		// write back to global mem
		image[(y * pitch) + x] = static_cast< unsigned char >(grayPix);
	}
#else
	// only run threads that are in the image
	if (x < width && y < height) {
		// execute grey scaling code without shared mem
		float grayPix = 0.0f;
		float r       = static_cast< float >(getElementRed(x, y));
		float g       = static_cast< float >(getElementGreen(x, y));
		float b       = static_cast< float >(getElementBlue(x, y));
		grayPix       = (0.3f * r) + (0.59f * g) + (0.11f * b);

		// write back to global mem
		image[(y * pitch) + x] = static_cast< unsigned char >(grayPix);
	}
#endif
}
#else
__global__ void rgb2grayCudaKernel(unsigned char *red, unsigned char *green, unsigned char *blue, unsigned int *data) {
	// load width and height
	unsigned int width  = getWidth(data);
	unsigned int height = getHeight(data);

	// load block ID's. Thread ids
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int redpitch   = getPitchRedImage(data);
	unsigned int greenpitch = getPitchGreenImage(data);
	unsigned int bluepitch  = getPitchBlueImage(data);

#ifdef SHARED_MEM
	// only run threads that are in the image
	if (x < width && y < height) {
		// allocate shared mem for this block
		__shared__ unsigned char redImage[MAX_BLOCKSIZE][MAX_BLOCKSIZE];
		__shared__ unsigned char greenImage[MAX_BLOCKSIZE][MAX_BLOCKSIZE];
		__shared__ unsigned char blueImage[MAX_BLOCKSIZE][MAX_BLOCKSIZE];

		// copy from global mem to shared mem
		redImage[threadIdx.x][threadIdx.y]   = red[(y * redpitch) + x];
		greenImage[threadIdx.x][threadIdx.y] = green[(y * greenpitch) + x];
		blueImage[threadIdx.x][threadIdx.y]  = blue[(y * bluepitch) + x];

		__syncthreads();

		// execute grey scaling code
		float grayPix = 0.0f;
		float r = static_cast< float >(redImage[threadIdx.x][threadIdx.y]);
		float g = static_cast< float >(greenImage[threadIdx.x][threadIdx.y]);
		float b = static_cast< float >(blueImage[threadIdx.x][threadIdx.y]);
		grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);

		__syncthreads();

		// write back to global mem
		red[(y * redpitch) + x] = static_cast< unsigned char >(grayPix);
	}
#else
	// only run threads that are in the image
	if (x < width && y < height) {
		// execute grey scaling code without shared mem
		float grayPix = 0.0f;
		float r = static_cast< float >(red[(y * redpitch) + x]);
		float g = static_cast< float >(green[(y * greenpitch) + x]);
		float b = static_cast< float >(blue[(y * bluepitch) + x]);
		grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);

		// write back to global mem
		red[(y * redpitch) + x] = static_cast< unsigned char >(grayPix);
	}
#endif
}
#endif

__host__ void rgb2gray(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height, double *totaltime) {
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	kernelTime.start();
	for ( int y = 0; y < height; y++ )
	{
		for ( int x = 0; x < width; x++ )
		{
			float grayPix = 0.0f;
			float r = static_cast< float >(inputImage[(y * width) + x]);
			float g = static_cast< float >(inputImage[(width * height) + (y * width) + x]);
			float b = static_cast< float >(inputImage[(2 * width * height) + (y * width) + x]);

			grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);

			grayImage[(y * width) + x] = static_cast< unsigned char >(grayPix);
		}
	}
	kernelTime.stop();
	*totaltime = kernelTime.getElapsed();
}

//-------------------------------------------------------------------------------------------------------------------
// histogram1D
//-------------------------------------------------------------------------------------------------------------------
#ifdef TEXTURE_MEM
__global__ void histogram1DCudaKernel(unsigned int *data) {
	// load width and height
	unsigned int width  = getWidth(data);
	unsigned int height = getHeight(data);

	// load block ID's. Thread ids
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int x = threadIdx.x + blockIdx.x * blockDim.x;

	// get histogram pointer
	unsigned int *histogram = getHistogram(data);
	unsigned int index;

#ifdef SHARED_MEM
	// only run threads that are in the image
	if (x < width && y < height) {
		// allocate shared mem for this block
		__shared__ unsigned char grayImage[MAX_BLOCKSIZE][MAX_BLOCKSIZE];

		// copy from global mem to shared mem
		grayImage[threadIdx.x][threadIdx.y] = getElementRed(x, y);

		__syncthreads();

		// get histogram index
		index = static_cast< unsigned int >(grayImage[threadIdx.x][threadIdx.y]);

		__syncthreads();

		// add pixel to histogram in one threadsafe operation
		atomicAdd(&histogram[index], 1);
	}
#else
	// only run threads that are in the image
	if (x < width && y < height) {
		// get histogram index
		index = static_cast< unsigned int >(getElementRed(x, y));

		// add pixel to histogram in one threadsafe operation
		atomicAdd(&histogram[index], 1);
	}
#endif
}
#else
__global__ void histogram1DCudaKernel(unsigned char* image, unsigned int *data) {
	// load width and height
	unsigned int width  = getWidth(data);
	unsigned int height = getHeight(data);

	// load block ID's. Thread ids
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pitch = getPitchRedImage(data);

	// get histogram pointer
	unsigned int *histogram = getHistogram(data);
	unsigned int index;

#ifdef SHARED_MEM
	// only run threads that are in the image
	if (x < width && y < height) {
		// allocate shared mem for this block
		__shared__ unsigned char grayImage[MAX_BLOCKSIZE][MAX_BLOCKSIZE];

		// copy from global mem to shared mem
		grayImage[threadIdx.x][threadIdx.y] = image[(y * pitch) + x];

		__syncthreads();

		// get histogram index
		index = static_cast< unsigned int >(grayImage[threadIdx.x][threadIdx.y]);

		__syncthreads();

		// add pixel to histogram in one threadsafe operation
		atomicAdd(&histogram[index], 1);
	}
#else
	// only run threads that are in the image
	if (x < width && y < height) {
		// get histogram index
		index = static_cast< unsigned int >(image[(y * pitch) + x]);

		// add pixel to histogram in one threadsafe operation
		atomicAdd(&histogram[index], 1);
	}
#endif
}
#endif

__host__ void histogram1D(unsigned char *grayImage, const int width, const int height,
                          unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, double *totaltime)
{
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	memset(reinterpret_cast< void * >(histogram), 0, HISTOGRAM_SIZE * sizeof(int));

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ )
	{
		for ( int x = 0; x < width; x++ )
		{
			histogram[static_cast< unsigned int >(grayImage[(y * width) + x])] += 1;
		}
	}
	// /Kernel
	kernelTime.stop();
	*totaltime = kernelTime.getElapsed();
}

//-------------------------------------------------------------------------------------------------------------------
// contrast1D
//-------------------------------------------------------------------------------------------------------------------
__global__ void contrastMinKernel(unsigned int *data) {
	// load width and height, histogramsize and contrast threshold
	unsigned int histogramSize     = getHistogramSize(data);
	unsigned int contrastThreshold = getContrastThreshold(data);

	// get histogram pointer
	unsigned int *histogram = getHistogram(data);

	// find minimum
	unsigned int i = 0;
	while ( (i < histogramSize) && (histogram[i] < contrastThreshold) ) {
		i++;
	}
	setMin(data, i);
}

__global__ void contrastMaxKernel(unsigned int *data) {
	// load width and height, histogramsize and contrast threshold
	unsigned int histogramSize     = getHistogramSize(data);
	unsigned int contrastThreshold = getContrastThreshold(data);
	unsigned int min               = getMin(data);

	// get histogram pointer
	unsigned int *histogram = getHistogram(data);

	// find maximum
	unsigned int i = histogramSize - 1;
	while ( (i > min) && (histogram[i] < contrastThreshold) ) {
		i--;
	}
	setMax(data, i);
}

#ifdef TEXTURE_MEM
__global__ void contrast1DKernel(unsigned char *image, unsigned int *data) {
	// load width and height
	unsigned int width  = getWidth(data);
	unsigned int height = getHeight(data);

	// load block ID's. Thread ids
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pitch = getPitchRedImage(data);

	// calculate difference
	float diff = getMax(data) - getMin(data);

	// get pixel
	unsigned int min = getMin(data);
	unsigned int max = getMax(data);

#ifdef SHARED_MEM
	// only run threads that are in the image
	if (x < width && y < height) {
		// allocate shared mem for this block
		__shared__ unsigned char grayImage[MAX_BLOCKSIZE][MAX_BLOCKSIZE];

		// copy from global mem to shared mem
		grayImage[threadIdx.x][threadIdx.y] = getElementRed(x, y);

		__syncthreads();

		// apply contrast enhancement
		unsigned char pixel = grayImage[threadIdx.x][threadIdx.y];
		if ( pixel < min )	{
			pixel = 0;
		}
		else if ( pixel > max )	{
			pixel = 255;
		}
		else	{
			pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
		}

		__syncthreads();

		// write back pixel
		image[(y * pitch) + x] = pixel;
	}
#else
	// only run threads that are in the image
	if (x < width && y < height) {
		// apply contrast enhancement
		unsigned char pixel = getElementRed(x, y);
		if ( pixel < min )	{
			pixel = 0;
		}
		else if ( pixel > max )	{
			pixel = 255;
		}
		else	{
			pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
		}

		// write back pixel
		image[(y * pitch) + x] = pixel;
	}
#endif
}
#else
__global__ void contrast1DKernel(unsigned char *image, unsigned int *data) {
	// load width and height
	unsigned int width  = getWidth(data);
	unsigned int height = getHeight(data);

	// load block ID's. Thread ids
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pitch = getPitchRedImage(data);

	// calculate difference
	float diff = getMax(data) - getMin(data);

	// get pixel
	unsigned int min = getMin(data);
	unsigned int max = getMax(data);

#ifdef SHARED_MEM
	// only run threads that are in the image
	if (x < width && y < height) {
		// allocate shared mem for this block
		__shared__ unsigned char grayImage[MAX_BLOCKSIZE][MAX_BLOCKSIZE];

		// copy from global mem to shared mem
		grayImage[threadIdx.x][threadIdx.y] = image[(y * pitch) + x];

		__syncthreads();

		// apply contrast enhancement
		unsigned char pixel = grayImage[threadIdx.x][threadIdx.y];
		if ( pixel < min )	{
			pixel = 0;
		}
		else if ( pixel > max )	{
			pixel = 255;
		}
		else	{
			pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
		}

		__syncthreads();

		// write back pixel
		image[(y * pitch) + x] = pixel;
	}
#else
	// only run threads that are in the image
	if (x < width && y < height) {
		// apply contrast enhancement
		unsigned char pixel = image[(y * pitch) + x];
		if ( pixel < min )	{
			pixel = 0;
		}
		else if ( pixel > max )	{
			pixel = 255;
		}
		else	{
			pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
		}

		// write back pixel
		image[(y * pitch) + x] = pixel;
	}
#endif
}
#endif

__host__ void contrast1D(unsigned char *grayImage, const int width, const int height,
                         unsigned int *histogram, const unsigned int HISTOGRAM_SIZE,
                         const unsigned int CONTRAST_THRESHOLD, double *totaltime)
{
	unsigned int i = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	while ( (i < HISTOGRAM_SIZE) && (histogram[i] < CONTRAST_THRESHOLD) )
	{
		i++;
	}
	unsigned int min = i;

	i = HISTOGRAM_SIZE - 1;
	while ( (i > min) && (histogram[i] < CONTRAST_THRESHOLD) )
	{
		i--;
	}
	unsigned int max = i;
	float diff = max - min;

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ )
	{
		for (int x = 0; x < width; x++ )
		{
			unsigned char pixel = grayImage[(y * width) + x];

			if ( pixel < min )
			{
				pixel = 0;
			}
			else if ( pixel > max )
			{
				pixel = 255;
			}
			else
			{
				pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
			}

			grayImage[(y * width) + x] = pixel;
		}
	}
	// /Kernel
	kernelTime.stop();
	*totaltime = kernelTime.getElapsed();
}

//-------------------------------------------------------------------------------------------------------------------
// triangularSmooth
//-------------------------------------------------------------------------------------------------------------------
#ifdef TEXTURE_MEM
__global__ void triangularSmoothKernel(unsigned char *image, unsigned int *data) {
	// load width and height
	unsigned int width  = getWidth(data);
	unsigned int height = getHeight(data);

	// load block ID's. Thread ids
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pitch = getPitchGreenImage(data);

	// only run threads that are in the image
	if (x < width && y < height) {
		// declare variables
		unsigned int filterItem = 0;
		float filterSum = 0.0f;
		float smoothPix = 0.0f;
		unsigned char value;
		int fy, fx;
		const float filter[] = {1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
		                        1.0f, 2.0f, 2.0f, 2.0f, 1.0f,
		                        1.0f, 2.0f, 3.0f, 2.0f, 1.0f,
		                        1.0f, 2.0f, 2.0f, 2.0f, 1.0f,
		                        1.0f, 1.0f, 1.0f, 1.0f, 1.0f
		                       };

		// do the smoothing
		for ( fy = y - 2; fy < y + 3; fy++ )
		{
			for ( fx = x - 2; fx < x + 3; fx++ )
			{
				if ( ((fy < 0) || (fy >= height)) || ((fx < 0) || (fx >= width)) )
				{
					filterItem++;
					continue;
				}

				smoothPix += getElementRed(fx, fy) * filter[filterItem];
				filterSum += filter[filterItem];
				filterItem++;
			}
		}
		smoothPix /= filterSum;
		value = static_cast< unsigned char >(smoothPix);

		// write back to global mem
		image[(y * pitch) + x] = value;
	}
}
#else
__global__ void triangularSmoothKernel(unsigned char *red, unsigned char *green, unsigned int *data) {
	// load width and height
	unsigned int width  = getWidth(data);
	unsigned int height = getHeight(data);

	// load block ID's. Thread ids
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int redpitch   = getPitchRedImage(data);
	unsigned int greenpitch = getPitchGreenImage(data);

	// only run threads that are in the image
	if (x < width && y < height) {
		// declare variables
		unsigned int filterItem = 0;
		float filterSum = 0.0f;
		float smoothPix = 0.0f;
		unsigned char value;
		int fy, fx;
		const float filter[] = {1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
		                        1.0f, 2.0f, 2.0f, 2.0f, 1.0f,
		                        1.0f, 2.0f, 3.0f, 2.0f, 1.0f,
		                        1.0f, 2.0f, 2.0f, 2.0f, 1.0f,
		                        1.0f, 1.0f, 1.0f, 1.0f, 1.0f
		                       };

		// do the smoothing
		for ( fy = y - 2; fy < y + 3; fy++ )
		{
			for ( fx = x - 2; fx < x + 3; fx++ )
			{
				if ( ((fy < 0) || (fy >= height)) || ((fx < 0) || (fx >= width)) )
				{
					filterItem++;
					continue;
				}

				smoothPix += red[(fy * redpitch) + fx] * filter[filterItem];
				filterSum += filter[filterItem];
				filterItem++;
			}
		}
		smoothPix /= filterSum;
		value = static_cast< unsigned char >(smoothPix);

		// write back to global mem
		green[(y * greenpitch) + x] = value;
	}
}
#endif

__host__ void triangularSmooth(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height, double *totaltime)
{
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ )
	{
		for ( int x = 0; x < width; x++ )
		{
			unsigned int filterItem = 0;
			float filterSum = 0.0f;
			float smoothPix = 0.0f;
			const float filter[] = {1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
			                        1.0f, 2.0f, 2.0f, 2.0f, 1.0f,
			                        1.0f, 2.0f, 3.0f, 2.0f, 1.0f,
			                        1.0f, 2.0f, 2.0f, 2.0f, 1.0f,
			                        1.0f, 1.0f, 1.0f, 1.0f, 1.0f
			                       };

			for ( int fy = y - 2; fy < y + 3; fy++ )
			{
				for ( int fx = x - 2; fx < x + 3; fx++ )
				{
					if ( ((fy < 0) || (fy >= height)) || ((fx < 0) || (fx >= width)) )
					{
						filterItem++;
						continue;
					}

					smoothPix += grayImage[(fy * width) + fx] * filter[filterItem];
					filterSum += filter[filterItem];
					filterItem++;
				}
			}

			smoothPix /= filterSum;
			smoothImage[(y * width) + x] = static_cast< unsigned char >(smoothPix);
		}
	}
	// /Kernel
	kernelTime.stop();
	*totaltime = kernelTime.getElapsed();
}

__host__ void imageProcess(unsigned char *inputImage, unsigned char *smoothImage, const int width, const int height,
                           const unsigned int HISTOGRAM_SIZE, const unsigned int CONTRAST_THRESHOLD, double *totaltime) {

//-------------------------------------------------------------------------------------------------------------------
// calculation occupancy GPU
//-------------------------------------------------------------------------------------------------------------------
	// variables
	unsigned int numBlocks[2];
	unsigned int numThreads[2];
	unsigned int widthCounter, heightCounter;

	// calculate number of blocks in both directions
	widthCounter = 1;
	while (width > numMultiProcessors * widthCounter * numThreadsPerBlock[0]) {
		widthCounter++;
	}
	numBlocks[0] = numMultiProcessors * widthCounter;
	heightCounter = 1;
	while (height > numMultiProcessors * heightCounter * numThreadsPerBlock[1]) {
		heightCounter++;
	}
	numBlocks[1] = numMultiProcessors * heightCounter;
	dim3 blockGrid(numBlocks[0], numBlocks[1]);

	// calculate number of threads per block in both directions
	numThreads[0] = width / numBlocks[0];
	if (width % numBlocks[0] > 0)
		numThreads[0]++;
	numThreads[1] = height / numBlocks[1];
	if (height % numBlocks[1] > 0)
		numThreads[1]++;
	dim3 threadGrid(numThreads[0], numThreads[1]);

//-------------------------------------------------------------------------------------------------------------------
// Write Memory to GPU device
//-------------------------------------------------------------------------------------------------------------------
	// device variables
	unsigned char *dev_redimage, *dev_greenimage, *dev_blueimage;
	unsigned int  *dev_data;
	// host variables
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	// initialize pitch values (will be filled in by the malloc function later)
	size_t redimage_pitch   = 0;
	size_t greenimage_pitch = 0;
	size_t blueimage_pitch  = 0;

	// allocate 2D memory on GPU
	if (hipMallocPitch<unsigned char>(&dev_redimage, &redimage_pitch, width, height) != hipSuccess) {
		cout << "imageProcess - cuda pitch malloc red failed" << endl;
		exit(1);
	}
	if (hipMallocPitch<unsigned char>(&dev_greenimage, &greenimage_pitch, width, height) != hipSuccess) {
		cout << "imageProcess - cuda pitch malloc green failed" << endl;
		exit(1);
	}
	if (hipMallocPitch<unsigned char>(&dev_blueimage, &blueimage_pitch, width, height) != hipSuccess) {
		cout << "imageProcess - cuda pitch malloc blue failed" << endl;
		exit(1);
	}
	if (hipMalloc(&dev_data, HISTOGRAM_SIZE * sizeof(int) + 9 * sizeof(int)) != hipSuccess) {
		cout << "imageProcess - cuda malloc data failed" << endl;
		exit(1);
	}
	unsigned int data[DATA_SIZE] = {width, height, HISTOGRAM_SIZE, CONTRAST_THRESHOLD, redimage_pitch, greenimage_pitch, blueimage_pitch};

	// start timing
	kernelTime.start();

	// copy data from host to device.
	if (hipMemcpy2D(dev_redimage, redimage_pitch, &inputImage[0], width, width, height, hipMemcpyHostToDevice) != hipSuccess) {
		cout << "imageProcess - cuda 2D mem cpy red failed" << endl;
		exit(1);
	}
	if (hipMemcpy2D(dev_greenimage, greenimage_pitch, &inputImage[width * height], width, width, height, hipMemcpyHostToDevice) != hipSuccess) {
		cout << "imageProcess - cuda 2D mem cpy green failed" << endl;
		exit(1);
	}
	if (hipMemcpy2D(dev_blueimage, blueimage_pitch, &inputImage[2 * width * height], width, width, height, hipMemcpyHostToDevice) != hipSuccess) {
		cout << "imageProcess - cuda 2D mem cpy blue failed" << endl;
		exit(1);
	}
	if (hipMemcpy(dev_data, data, DATA_SIZE * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
		cout << "imageProcess - cuda 2D mem cpy data failed" << endl;
		exit(1);
	}
	if (hipMemset(&dev_data[9], 0, HISTOGRAM_SIZE * sizeof(int)) != hipSuccess) {
		cout << "imageProcess - cuda mem set histogram failed" << endl;
		exit(1);
	}

	// stop timing
	kernelTime.stop();
	totaltime[0] = kernelTime.getElapsed();

#ifdef TEXTURE_MEM
	//Bind the image to the texture. Now the kernel will read the input image through the texture cache.
	if (hipBindTexture2D(NULL, texture2DRed, dev_redimage, width, height, redimage_pitch) != hipSuccess) {
		cout << "imageProcess - cuda bind 2D texture red failed" << endl;
		exit(1);
	}
	if (hipBindTexture2D(NULL, texture2DGreen, dev_greenimage, width, height, greenimage_pitch) != hipSuccess) {
		cout << "imageProcess - cuda bind 2D texture green failed" << endl;
		exit(1);
	}
	if (hipBindTexture2D(NULL, texture2DBlue, dev_blueimage, width, height, blueimage_pitch) != hipSuccess) {
		cout << "imageProcess - bind 2D texture blue failed" << endl;
		exit(1);
	}

	// set border access to zero
	texture2DRed.addressMode[0]   = texture2DRed.addressMode[1] 	= hipAddressModeBorder;
	texture2DGreen.addressMode[0] = texture2DGreen.addressMode[1] 	= hipAddressModeBorder;
	texture2DBlue.addressMode[0]  = texture2DBlue.addressMode[1] 	= hipAddressModeBorder;
#endif

//-------------------------------------------------------------------------------------------------------------------
// RGB to gray scale conversion
//-------------------------------------------------------------------------------------------------------------------
	// start timing
	kernelTime.reset();
	kernelTime.start();

	// create grid for kernel functions, execute the kernel on the GPU
#ifdef TEXTURE_MEM
	rgb2grayCudaKernel <<< blockGrid, threadGrid>>>(dev_redimage, dev_data);
#else
	rgb2grayCudaKernel <<< blockGrid, threadGrid>>>(dev_redimage, dev_greenimage, dev_blueimage, dev_data);
#endif
	if (hipGetLastError() != hipSuccess) {
		cout << "imageProcess - cuda start rgb2gray kernel on device failed" << endl;
		exit(1);
	}

	// stop timing
	kernelTime.stop();
	totaltime[1] = kernelTime.getElapsed();

//-------------------------------------------------------------------------------------------------------------------
// Creating histogram
//-------------------------------------------------------------------------------------------------------------------
	// start timing
	kernelTime.reset();
	kernelTime.start();

	// create grid for kernel functions, execute the kernel on the GPU
#ifdef TEXTURE_MEM
	histogram1DCudaKernel <<< blockGrid, threadGrid>>>(dev_data);
#else
	histogram1DCudaKernel <<< blockGrid, threadGrid>>>(dev_redimage, dev_data);
#endif
	if (hipGetLastError() != hipSuccess) {
		cout << "imageProcess - cuda start histogram1D kernel on device failed" << endl;
		exit(1);
	}

	// stop timing
	kernelTime.stop();
	totaltime[2] = kernelTime.getElapsed();

//-------------------------------------------------------------------------------------------------------------------
// Computing contrast
//-------------------------------------------------------------------------------------------------------------------
	// start timing
	kernelTime.reset();
	kernelTime.start();

	// create grid for kernel functions, execute the kernel on the GPU
	contrastMinKernel <<< 1, 1>>>(dev_data);
	if (hipGetLastError() != hipSuccess) {
		cout << "imageProcess - cuda start contrastMin kernel on device failed" << endl;
		exit(1);
	}

	// create grid for kernel functions, execute the kernel on the GPU
	contrastMaxKernel <<< 1, 1>>>(dev_data);
	if (hipGetLastError() != hipSuccess) {
		cout << "imageProcess - cuda start contrastMax kernel on device failed" << endl;
		exit(1);
	}

	// create grid for kernel functions, execute the kernel on the GPU
#ifdef TEXTURE_MEM
	contrast1DKernel <<< blockGrid, threadGrid>>>(dev_redimage, dev_data);
#else
	contrast1DKernel <<< blockGrid, threadGrid>>>(dev_redimage, dev_data);
#endif
	if (hipGetLastError() != hipSuccess) {
		cout << "imageProcess - cuda start contrast1D kernel on device failed" << endl;
		exit(1);
	}

	// stop timing
	kernelTime.stop();
	totaltime[3] = kernelTime.getElapsed();

//-------------------------------------------------------------------------------------------------------------------
// Computing Smoothing
//-------------------------------------------------------------------------------------------------------------------
	// start timing
	kernelTime.reset();
	kernelTime.start();

	// create grid for kernel functions, execute the kernel on the GPU
#ifdef TEXTURE_MEM
	triangularSmoothKernel <<< blockGrid, threadGrid>>>(dev_greenimage, dev_data);
#else
	triangularSmoothKernel <<< blockGrid, threadGrid>>>(dev_redimage, dev_greenimage, dev_data);
#endif
	if (hipGetLastError() != hipSuccess) {
		cout << "triangularSmoothCuda - cuda start kernels on device failed" << endl;
		exit(1);
	}

	// stop timing
	kernelTime.stop();
	totaltime[4] = kernelTime.getElapsed();

	// start timing
	kernelTime.reset();
	kernelTime.start();

	// read back result from GPU
	if (hipMemcpy2D(smoothImage, width, dev_greenimage, greenimage_pitch, width, height, hipMemcpyDeviceToHost) != hipSuccess) {
		cout << "triangularSmoothCuda - cuda mem copy smooth image to host failed" << endl;
		exit(1);
	}

	// stop timing
	kernelTime.stop();
	totaltime[5] = kernelTime.getElapsed();

#ifdef TEXTURE_MEM
	//Release the texture
	hipUnbindTexture(texture2DRed);
	hipUnbindTexture(texture2DGreen);
	hipUnbindTexture(texture2DBlue);
#endif

	// free memory on GPU
	hipFree(dev_redimage);
	hipFree(dev_greenimage);
	hipFree(dev_blueimage);
	hipFree(dev_data);
}